
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hipblas.h>


int  main(int argc, char**argv)
{
    
	const int  rows = 3, cols = 2;

    //size in bytes
    const int ARRAY_BYTES = ( rows * cols ) * sizeof(int);
    
	float *A;
	A = (float *) malloc(ARRAY_BYTES);
	
	//initialize
	A[ 0 ] = 0;
	A[ 1 ] = 1;
	A[ 2 ] = 2;
	A[ 3 ] = 3;
	A[ 4 ] = 4;
	A[ 5 ] = 5;
			
	// print matrix
	printf("\nA matrix");
	for (int i = 0; i < rows; ++i) {
		for (int j = 0; j < cols; ++j)
			printf("\nA = %f",A[ i + rows * j ]);
		printf("\n");
	}

	float *A_dev , *C_dev;
	hipMalloc((void **) &A_dev, ARRAY_BYTES);
	hipMalloc((void **) &C_dev, ARRAY_BYTES);
	
	hipMemcpy(A_dev, A, ARRAY_BYTES, hipMemcpyHostToDevice);

	float const alpha(1.0);
    float const beta(0.0);
    
	hipblasHandle_t handle;

	hipblasStatus_t status;

	status = hipblasCreate(&handle);
	
	//use cublasSetPointerMode  HOST in order to be able to use alpha and beta in host ,else you must define them in device
	hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST); 
	status = hipblasSgeam(handle,HIPBLAS_OP_T, HIPBLAS_OP_N,  cols, rows , &alpha ,A_dev ,rows , &beta ,A_dev ,rows, C_dev , cols);

 	hipMemcpy(A,C_dev, ARRAY_BYTES, hipMemcpyDeviceToHost);

	printf("\nA transposed ");
	for (int  i = 0; i < cols; ++i) {
		for (int j = 0; j < rows; ++j)
			printf("\nA = %f", A[ i + cols * j ]);
		printf("\n");
	}
  
	hipFree(A_dev);
	hipFree(C_dev);
	
	free(A);
	
    return 0;
}


