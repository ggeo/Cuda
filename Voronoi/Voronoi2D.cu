#include <assert.h>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <cstring>
#include <ctime>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <float.h>
#include <cmath>


/*
 *	This macro checks for API errors in the CUDA calls.
 */

#define gpuErrchk(ans) { gpuAssert( (ans), __FILE__, __LINE__ ); }

inline void
gpuAssert( hipError_t code, const char * file, int line, bool abort = true )
{
	if ( hipSuccess != code )
	{
		fprintf( stderr, "\nGPUassert: %s %s %d\n", hipGetErrorString( code ), file, line );
		if ( abort )
			exit( code );
	}


	return;

} /* gpuAssert */

/* ========================================================================== */
/*   Voronoi2D                                                                 */
/* -------------------------------------------------------------------------- */
/*!
 * @function    Voronoi2D
 *
 * @abstract
 *
 * @discussion  Calculates Voronoi cells
 *
 * @param	inNbOfSites [input] The number of the sites (seeds).
 * 								type: const size_t
 *
 * @param   inWidth [input] The width of the Voronoi image.
 *							type: const size_t
 *					
 * @param   inHeight [input] The height of the Voronoi image.
 * 							 type: const size_t
 * 							
 * @param	inX [input] The x coordinates of the points
 *						Dimensions :  Nx , type: float
 * 
 * @param	inY [input] The y coordinates of the points
 *						Dimensions :  Ny , type: float		
 * 				  
 * @param   inV [input] The inV holds for applying a threshold/color
 * 						to the cell region
 *						Dimensions : inNbOfSites, type: int
 * 		    
 * @param	ouVoronoi [output] The output data (pixels)
 *          Dimensions :  The total number of threads in the grid
 *          ( theBlocksPerGridX * theBlocksPerGridY * theThreadsPerBlockX * theThreadsPerBlockY )
 *			type: float
 */
 /* ========================================================================== */

__global__ void Voronoi2D(

const   size_t  inNbOfSites,
	const   size_t  inWidth,
	const   size_t  inHeight,
	float *  const  inX,
	float *  const  inY,
	int   *  const  inV,
	int   *  const  ouVoronoi )
	{
		
		float distX , distY;
		float theTempDistance ,theDistance = FLT_MAX;
		int theThreshold;
		
		//loop through all points calculating distance
		for ( int y = ( ( blockIdx.y * blockDim.y ) + threadIdx.y ); y < inHeight; y += blockDim.y * gridDim.y )
		{
			for ( int x = ( ( blockIdx.x * blockDim.x ) + threadIdx.x ); x < inWidth; x += blockDim.x * gridDim.x )
			{
				int theGlobalIdx = y * ( blockDim.x * gridDim.x ) + x;
					
				//Calculate distances for all the points
				for ( int i = 0; i < inNbOfSites; i++ )
				{
						
					distX = inX[ i ] - x;
					distY = inY[ i ] - y;
						
					theTempDistance = distX * distX + distY * distY;
						
					//if this Point is closer , assign proper threshold
					if ( theTempDistance < theDistance )
					{
						theDistance = theTempDistance;
						theThreshold = inV[ i ];
					}
				}
					
				//write result back to global memory
				*( ouVoronoi + theGlobalIdx ) = theThreshold;	
					
			} /* x */
				
		} /* y */
								
}
	
int main()
{
	const size_t Width = 256 , Height = 256;
	const size_t Nx = 128 , Ny = 128;
	const size_t NbOfSites = 100; //should be <= Nx and Ny
	const size_t ThreadsPerBlockX = 16 , ThreadsPerBlockY = 16 ,BlocksPerGridX = Width / 16 , BlocksPerGridY = Height / 16;
	const size_t TotalNbOfPixels = ( Width * Height );
	
	// Allocate host memory
	float * X = (float*) malloc( Nx * sizeof (*X) );
	assert( NULL != X );
	float * Y = (float*) malloc( Ny * sizeof (*Y) );
	assert( NULL != Y );
	int * V = (int*) malloc( NbOfSites * sizeof (*V) );
	assert( NULL != V );
	int * VoronoiDiagram = (int*) malloc ( TotalNbOfPixels * sizeof(*VoronoiDiagram) );
	assert( NULL != VoronoiDiagram );
	
	float * devX , * devY;
	int * devVoronoiDiagram , * devV;
	// Allocate device memory
	gpuErrchk( hipMalloc( (void**) &devX, Nx * sizeof(*devX) ) );
	gpuErrchk( hipMalloc( (void**) &devY, Ny * sizeof(*devY) ) );
	gpuErrchk( hipMalloc( (void**) &devV, NbOfSites * sizeof(*devV) ) );
	gpuErrchk( hipMalloc( (void**) &devVoronoiDiagram,  TotalNbOfPixels * sizeof(*devVoronoiDiagram) ) );
	
	// Create random coordinates
	srand((unsigned int)time(NULL));
	for ( int i = 0; i < Nx; i++ )	X[ i ] = ( ( (float) rand() / (float) ( RAND_MAX ) ) * Width );
	for ( int i = 0; i < Ny; i++ )  Y[ i ] = ( ( (float) rand() / (float) ( RAND_MAX ) ) * Height );
	
	for ( int i = 0; i < NbOfSites; i++ )	V[ i ] = i;
	
	// Define grid dimensions
	dim3 BlocksDim ( BlocksPerGridX , BlocksPerGridY );
	dim3 ThreadsPerBlock ( ThreadsPerBlockX , ThreadsPerBlockY );
	
	gpuErrchk( hipMemcpy( devV , V , NbOfSites * sizeof( *V ), hipMemcpyHostToDevice ) );
	gpuErrchk( hipMemcpy( devX , X , Nx * sizeof( *X ), hipMemcpyHostToDevice ) );
	gpuErrchk( hipMemcpy( devY , Y , Ny * sizeof( *Y ), hipMemcpyHostToDevice ) );
		
  	hipEvent_t CurrentEventPre,
  	CurrentEventPost;
  	float CurrentPostPreTimeMS;
  
  	gpuErrchk( hipEventCreate( &CurrentEventPre ) );
  	gpuErrchk( hipEventCreate( &CurrentEventPost ) );
  
  	gpuErrchk( hipEventRecord( CurrentEventPre ) );
  
  
  	Voronoi2D<<< BlocksDim,ThreadsPerBlock >>>( NbOfSites, 
  												Width,
  												Height,
  												devX,
  												devY,
  												devV,
  												devVoronoiDiagram );
  
  	gpuErrchk( hipPeekAtLastError() );
  	gpuErrchk( hipDeviceSynchronize() );
  
  	gpuErrchk( hipEventRecord( CurrentEventPost ) );
  	gpuErrchk( hipEventSynchronize( CurrentEventPost ) );
  	gpuErrchk( hipEventElapsedTime( &CurrentPostPreTimeMS, CurrentEventPre, CurrentEventPost ) );
  	printf( "\nGPU time for calling Voronoi: %f ms\n", CurrentPostPreTimeMS );
  
  	gpuErrchk( hipMemcpy( VoronoiDiagram,
  						   devVoronoiDiagram , 
  						   TotalNbOfPixels * sizeof(*devVoronoiDiagram), hipMemcpyDeviceToHost ) );
  
  	{
		FILE * theFile;		
	  	theFile = fopen( "Voronoi2D", "wb" );
	  	assert( NULL != theFile );
	  	assert( TotalNbOfPixels == fwrite( VoronoiDiagram , sizeof(*devVoronoiDiagram), TotalNbOfPixels , theFile ) );
	  	fclose( theFile );
  	}
  
  	//free memory
  	gpuErrchk( hipFree( devX ) );
  	gpuErrchk( hipFree( devY ) );
  	gpuErrchk( hipFree( devV ) );
  	gpuErrchk( hipFree( devVoronoiDiagram ) );
  
  	free( X );
  	free( Y );
  	free( V );
  	free( VoronoiDiagram );
  
  	return 0;
}
						